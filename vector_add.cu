
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    const int n = 10;
    int a[n], b[n], c[n];
    int *d_a, *d_b, *d_c;

    // Allocate memory on device
    hipMalloc(&d_a, n * sizeof(int));
    hipMalloc(&d_b, n * sizeof(int));
    hipMalloc(&d_c, n * sizeof(int));

    // Initialize host arrays
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = n - i;
    }

    // Copy input data to device
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    vectorAdd<<<1, n>>>(d_a, d_b, d_c, n);

    // Copy output data to host
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < n; i++) {
        // if (c[i] != n) {
        //     printf("Error: c[%d] = %d\n", i, c[i]);
        //     return 1;
        // }
        printf("c[%d] = %d", i, c[i]);
    }

    printf("Success!\n");

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
